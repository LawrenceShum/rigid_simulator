#include "hip/hip_runtime.h"
# pragma once
# include "rigid_body.cuh"
# include "rigid_geometry.cuh"
# include <sstream>
# include <iostream>


void Rigid_Body::determineLayout(dim3& gridLayout, dim3& blockLayout, int num)
{
	if (num <= nThreadMax_perBlock)
	{
		gridLayout = 1;
		blockLayout = num;
	}
	else
	{
		int split = num / nThreadMax_perBlock;
		gridLayout = split + 1;
		blockLayout = nThreadMax_perBlock;
	}
}

void Rigid_Geometry::load_mesh(const std::string file_name)
{
	std::ifstream in;
	in.open(file_name, std::ifstream::in);
	if (in.fail()) return;
	std::string line;
	while (!in.eof())
	{
		std::getline(in, line);
		std::istringstream iss(line.c_str());
		char trash;
		if (!line.compare(0, 2, "v "))
		{
			iss >> trash;
			double x, y, z;
			iss >> x >> y >> z;
			vertices.push_back(x);
			vertices.push_back(y);
			vertices.push_back(z);
		}
		else if (!line.compare(0, 2, "f "))
		{
			int f, t, n;
			iss >> trash;
			iss >> f >> t >> n;
			face.push_back(f - 1);
			face.push_back(t - 1);
			face.push_back(n - 1);
		}
	}
	in.close();
	num_of_vertices = vertices.size() / 3;
	copy_to_device();
	std::cout << "total vertices = " << num_of_vertices << ", faces = " << face.size() / 3 << std::endl;
}

void Rigid_Geometry::copy_to_device()
{
	checkCudaErrors(hipMalloc((void**)&vertices_in_gpu, sizeof(double) * num_of_vertices * 3));
	checkCudaErrors(hipMalloc((void**)&vertices_in_gpu_this_step, sizeof(double) * num_of_vertices * 3));
	checkCudaErrors(hipMemcpy(vertices_in_gpu, vertices.data(), sizeof(double) * num_of_vertices * 3, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(vertices_in_gpu_this_step, vertices.data(), sizeof(double) * num_of_vertices * 3, hipMemcpyHostToDevice));
}

int Rigid_Geometry::get_num_of_vertices()
{
	return num_of_vertices;
}

double* Rigid_Geometry::get_vertices_device()
{
	return vertices_in_gpu;
}

double* Rigid_Geometry::get_vertices_cpu()
{
	return vertices.data();
}

__global__ void update_vertices_gpu(double3 x, double* rotate_matrix, double* vertices, double* output, int num)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < num)
	{
		double3 vertex = make_type3(vertices[3 * index + 0], vertices[3 * index + 1], vertices[3 * index + 2]);
		//����ת
		double3 after_rotate;
		after_rotate.x = rotate_matrix[0] * vertex.x + rotate_matrix[1] * vertex.y + rotate_matrix[2] * vertex.z;
		after_rotate.y = rotate_matrix[3] * vertex.x + rotate_matrix[4] * vertex.y + rotate_matrix[5] * vertex.z;
		after_rotate.z = rotate_matrix[6] * vertex.x + rotate_matrix[7] * vertex.y + rotate_matrix[8] * vertex.z;
		double3 after_translation;
		after_translation.x = after_rotate.x + x.x;
		after_translation.y = after_rotate.y + x.y;
		after_translation.z = after_rotate.z + x.z;
		output[3 * index + 0] = after_translation.x;
		output[3 * index + 1] = after_translation.y;
		output[3 * index + 2] = after_translation.z;
	}
}

void Rigid_Geometry::update_vertices_location(double3 X, quaternion Q)
{
	dim3 gridLayout, blockLayout;
	if (num_of_vertices <= nThreadMax_perBlock)
	{
		gridLayout = 1;
		blockLayout = num_of_vertices;
	}
	else
	{
		gridLayout = (num_of_vertices / nThreadMax_perBlock) + 1;
		blockLayout = nThreadMax_perBlock;
	}
	double* R = new double[9];
	quaternion_to_matrix(Q, R);
	double* matrix_temp;
	checkCudaErrors(hipMalloc((void**)&matrix_temp, sizeof(double) * 9));
	checkCudaErrors(hipMemcpy(matrix_temp, R, sizeof(double) * 9, hipMemcpyHostToDevice));
	update_vertices_gpu << <gridLayout, blockLayout >> >
		(X, matrix_temp, vertices_in_gpu, vertices_in_gpu_this_step, num_of_vertices);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void Rigid_Geometry::copy_to_opengl_buffer(double* opengl_buffer)
{
	checkCudaErrors(hipMemcpy(opengl_buffer, vertices_in_gpu_this_step, sizeof(double) * num_of_vertices * 3, hipMemcpyDeviceToHost));
}

void Rigid_Body::set_timestep(double timestep)
{
	this->dt = timestep;
}

/*****************************translational_move*****************************/
void Rigid_Body::compute_translational_force()
{
	int num_vertices = geometry->get_num_of_vertices();
	translational_force.x = 0.0;
	translational_force.y = 0.0;
	translational_force.z = 0.0;
	for (int i = 0; i < num_vertices; i++)
	{
		double3 temp = forces_cpu[i];
		translational_force.x += temp.x;
		translational_force.y += temp.y;
		translational_force.z += temp.z;
	}
}

void Rigid_Body::translational_move()
{
	mass = 10;
	dim3 gridLayout, blockLayout;
	//��������
	double gravity_acceleration = -9.8;
	velocity.x += dt * (1.0 / mass) * translational_force.x;
	velocity.y += dt * ((1.0 / mass) * translational_force.y);
	velocity.z += dt * (1.0 / mass) * translational_force.z;
	X.x += dt * velocity.x;
	X.y += dt * velocity.y;
	X.z += dt * velocity.z;
}
/**************************************************************************/

/*******************************��������*******************************/
__global__ void compute_torque_elements_gpu(
	double* torque_elements, 
	double3* force_element,
	double* reference_vertices,
	double* rotate_matrix,
	int num)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	double3 vertex = make_type3(reference_vertices[index * 3 + 0], reference_vertices[index * 3 + 1], reference_vertices[index * 3 + 2]);
	if (index < num)
	{
		double3 torque;
		double3 force = force_element[index];
		double3 left;
		left.x = rotate_matrix[0] * vertex.x + rotate_matrix[1] * vertex.y + rotate_matrix[2] * vertex.z;
		left.y = rotate_matrix[3] * vertex.x + rotate_matrix[4] * vertex.y + rotate_matrix[5] * vertex.z;
		left.z = rotate_matrix[6] * vertex.x + rotate_matrix[7] * vertex.y + rotate_matrix[8] * vertex.z;
		torque.x = left.y * force.z - left.z * force.y;
		torque.y = left.x * force.z - left.z * force.x;
		torque.z = left.x * force.y - left.y * force.x;
		torque_elements[3 * index + 0] = torque.x;
		torque_elements[3 * index + 1] = torque.y;
		torque_elements[3 * index + 2] = torque.z;
	}
}

void Rigid_Body::compute_torque()
{
	int num_vertices = geometry->get_num_of_vertices();
	double* reference_vertices = geometry->get_vertices_device();
	double* temp_R;
	checkCudaErrors(hipMalloc((void**)&temp_R, sizeof(double) * 9));
	checkCudaErrors(hipMemcpy(temp_R, Rotate_Matrix, sizeof(double) * 9, hipMemcpyHostToDevice));

	//thrust::device_vector<double3> torque_element(num_vertices);
	double* torque_element;
	checkCudaErrors(hipMalloc((void**)&torque_element, sizeof(double) * num_vertices * 3));

	dim3 gridLayout, blockLayout;
	determineLayout(gridLayout, blockLayout, num_vertices);
	compute_torque_elements_gpu << <gridLayout, blockLayout >> >
		(torque_element, forces_gpu, reference_vertices, temp_R, num_vertices);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	double* tt = new double[num_vertices * 3];
	checkCudaErrors(hipMemcpy(tt, torque_element, sizeof(double) * 3 * num_vertices, hipMemcpyDeviceToHost));
	double3 sum_of_torque = make_type3(0.0, 0.0, 0.0);
	for (int i = 0; i < num_vertices; i++)
	{
		sum_of_torque.x += tt[i * 3 + 0];
		sum_of_torque.y += tt[i * 3 + 1];
		sum_of_torque.z += tt[i * 3 + 2];
	}
	T.x = sum_of_torque.x;
	T.y = sum_of_torque.y;
	T.z = sum_of_torque.z;
	checkCudaErrors(hipFree(temp_R));
	checkCudaErrors(hipFree(torque_element));
}
/***********************************************************************/

void Rigid_Body::compute_force()
{
	int num = geometry->get_num_of_vertices();
	for (int i = 0; i < num; i++)
	{
		forces_cpu[i].x = 0.0;
		forces_cpu[i].y = 0.0;
		forces_cpu[i].z = 0.0;
	}
	checkCudaErrors(hipMemcpy(forces_gpu, forces_cpu, sizeof(double3) * num, hipMemcpyHostToDevice));
}

void Rigid_Body::rotational_move()
{
	Eigen::Matrix3d RR;
	RR(0, 0) = Rotate_Matrix[0];	RR(0, 1) = Rotate_Matrix[1];	RR(0, 2) = Rotate_Matrix[2];
	RR(1, 0) = Rotate_Matrix[3];	RR(1, 1) = Rotate_Matrix[4];	RR(1, 2) = Rotate_Matrix[5];
	RR(2, 0) = Rotate_Matrix[6];	RR(2, 1) = Rotate_Matrix[7];	RR(2, 2) = Rotate_Matrix[8];
	Eigen::Matrix3d II;
	II(0, 0) = inertia[0];	II(0, 1) = inertia[1];	II(0, 2) = inertia[2];
	II(1, 0) = inertia[3];	II(1, 1) = inertia[4];	II(1, 2) = inertia[5];
	II(2, 0) = inertia[6];	II(2, 1) = inertia[7];	II(2, 2) = inertia[8];
	Eigen::Matrix3d right = II * RR.transpose();
	Eigen::Matrix3d I = RR * right;
	Eigen::Vector3d TT(T.x, T.y, T.z);
	Eigen::Vector3d mid = dt * I.inverse() * TT;
	angular_velocity.x += mid[0];
	//angular_velocity.y += mid[1];
	angular_velocity.y = 2;
	angular_velocity.z += mid[2];
	Q = Q + make_quaternion(0.0, 0.5 * dt * angular_velocity) * Q;
}

void Rigid_Body::compute_inertia()
{
	int num = geometry->get_num_of_faces();
	double* vertices = geometry->get_vertices_cpu();
	double vertex_weight = 0.01;
	Eigen::Matrix3d II = Eigen::Matrix3d::Zero();
	for (int i = 0; i < num; i++)
	{
		Eigen::Vector3d r_i(vertices[3 * i + 0], vertices[3 * i + 1], vertices[3 * i + 2]);
		II += vertex_weight * (r_i.dot(r_i) * Eigen::Matrix3d::Identity() - r_i * r_i.transpose());
	}
	for (int row = 0; row < 3; row++)
	{
		for (int col = 0; col < 3; col++)
		{
			inertia[row * 3 + col] = II(row, col);
		}
	}
}

void Rigid_Body::step_forward(double dt)
{
	set_timestep(dt);
	quaternion_to_matrix(Q, Rotate_Matrix);
	compute_force();
	compute_translational_force();
	translational_move();
	compute_torque();
	rotational_move();
	geometry->update_vertices_location(X, Q);
}